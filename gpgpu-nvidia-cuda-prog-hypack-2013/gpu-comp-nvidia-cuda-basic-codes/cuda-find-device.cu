
/*************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                       October 15-18, 2013

  Example     :  cuda-find-device.cu
 
  Objective   : Write a CUDA  program to set the gpu .                 

  Input       : None 

  Output      : information about device. 

  Created     : August-2013

  E-mail      : hpcfte@cdac.in     

***********************************************************************/

#include <stdio.h> 
#include <time.h> 
#include <hip/hip_runtime.h> 

/* Utility Macro : CUDA SAFE CALL */
void CUDA_SAFE_CALL( hipError_t call)
{

    hipError_t ret = call;
    switch(ret)
    {

        case hipSuccess:
              break;
        default :
              {

            printf(" ERROR at line :%i.%d' ' %s\n",
            __LINE__,ret,hipGetErrorString(ret));
            exit(-1);
            break;

             }

    }

}

int main ( void ) {

    int count;
    int dev;
    hipDeviceProp_t prop;

    CUDA_SAFE_CALL(hipGetDeviceCount( &count) );

    for(int i = 0; i < count; i++) {

        CUDA_SAFE_CALL( hipGetDeviceProperties( &prop, i) );

        CUDA_SAFE_CALL( hipGetDevice(&dev) );

        printf("Information about the device \t: %d\n", count);

        printf("Name \t\t\t\t: %s\n",prop.name);

        printf("ID of the device : %d\n", dev);
	memset(&prop, 0, sizeof (hipDeviceProp_t));

        prop.major = 1;
        prop.minor = 3;
        CUDA_SAFE_CALL( hipChooseDevice(&dev, &prop ) );

        printf("ID of CUDA device closest to revision 1.3 :%d \n", dev);

        CUDA_SAFE_CALL( hipSetDevice(dev) );

    }
    return 0;

}
