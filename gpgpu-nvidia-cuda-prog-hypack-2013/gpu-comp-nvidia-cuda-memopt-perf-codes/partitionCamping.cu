#include "hip/hip_runtime.h"


/************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

       Source Code  : partitionCamping.cu  

       Objective    : To demonstrate the difference in bandwidth achieved when    
                      blocks access global memory with and without partition     
                      camping. This Program measures the bandwidth of global 
                      memory for the initialization operation [a(i) = value] 
                      using NVIDIA GPU

	Input	    : None 

        Output      : Bandwidth achieved and timing (average) 

       Created      : August-2013

      E-mail        : hpcfte@cdac.in     

*************************************************************************/

#include <stdio.h>
#include <hip/hip_runtime.h>

#define ARRAY_SIZE 2195264
#define BLOCK_SIZE 64
#define NTIMES 2
#define HLINE "--------------------------------------------------------------\n"

void printResults();
void printDeviceDetails();
void cudaSafeMalloc(void ** , size_t );
void CudaGetDeviceProperties(hipDeviceProp_t *, int);
void CudaGetDevice(int *);
void checkCudaErrors();


float avgTime[2] = {0};
char *label[] = {"Without Partition Camping","With Partition Camping   "};

///////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for initializing the array without Partition Camping
///////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initializationWithoutPartitionCamping(float *array,  float value, int size)
{
	int idx = threadIdx.x + blockIdx.x * blockDim.x;
	if (idx < size)
		array[idx] = value;
}

///////////////////////////////////////////////////////////////////////////////////////////////////////



///////////////////////////////////////////////////////////////////////////////////////////////////////
// Kernel for initializing the array with Partition Camping
///////////////////////////////////////////////////////////////////////////////////////////////////////

__global__ void initializationWithPartitionCamping(float *array,  float value, int size)
{
	int blockIdx_x;
	blockIdx_x =  (blockIdx.x * 6) % gridDim.x;
	int idx = threadIdx.x + blockIdx_x * blockDim.x;
	if (idx < size)
		array[idx] = value;
}



int main()
{
	float *d_array;
	size_t size = ARRAY_SIZE *	sizeof(float);
	int i,j;	
	float elapsedTimes[2][NTIMES];
	hipEvent_t start,stop;

	
	// event creation, which will be used for timing the code
	hipEventCreate(&start);
	hipEventCreate(&stop);

	cudaSafeMalloc((void **) &d_array, size);

	int gridSize = ARRAY_SIZE / BLOCK_SIZE;
	if(ARRAY_SIZE % BLOCK_SIZE != 0) gridSize += 1;
	dim3 grid, block;
	block.x = BLOCK_SIZE;
	grid.x = gridSize;

	for(i=0; i<NTIMES; i++)
	{

	   // timing the initialization without Partition Camping
		hipEventRecord(start,0);
		initializationWithoutPartitionCamping<<<grid, block>>>(d_array, 1.0f, ARRAY_SIZE);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTimes[0][i],start,stop);
		checkCudaErrors();


	   // timing the initialization with Partition Camping
		hipEventRecord(start,0);
		initializationWithPartitionCamping<<< grid, block>>>(d_array, 1.0f, ARRAY_SIZE);
		hipEventRecord(stop,0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&elapsedTimes[1][i],start,stop);
		checkCudaErrors();
	}


	//Computing average time taken
	for(i=0; i<2; i++)
	{
		for(j=1; j<NTIMES; j++) //skipping first iteration
		{
			avgTime[i] += elapsedTimes[i][j];
		}
		avgTime[i] = avgTime[i]/(NTIMES-1);
	}
	
	// Printing the results 
	printResults();
	
	hipEventDestroy(start);
	hipEventDestroy(stop);

	return 0;
}





////////////////////////////////////////////////////////////////////////////////////////////////////////////
//
// Host Function to print the results
//
/////////////////////////////////////////////////////////////////////////////////////////////////////////////

void printResults()
{
	int j;
	printf("\n\n");
	printf(HLINE);
	printf("PARTITION CAMPING DEMONSTRATION\n");
	printf(HLINE);
	printDeviceDetails();
	printf(HLINE);
	printf("Array Size = %llu\n",(unsigned long long)ARRAY_SIZE);
	printf("Block Size = %d\n",(int)BLOCK_SIZE);
	printf(HLINE);
 
	printf("Initialization               Rate (GB/s)   Avg time  \n");
   for (j=0; j<2; j++) 
	{
		printf("%s%11.4f  %11.4f  \n", label[j], 1.0E-06 * (ARRAY_SIZE * sizeof(float))/avgTime[j],avgTime[j]);
	} 
	
	printf(HLINE);
}

void printDeviceDetails()
{
	int deviceId;
	hipDeviceProp_t prop;
	CudaGetDevice(&deviceId);
	 
	CudaGetDeviceProperties(&prop, deviceId);
	
	printf("Device Name is %s\n", prop.name);
	//printf("Clock Rate of this device is %f GHz\n",(float)prop.clockRate * 1.0E-06);
	printf("Compute Capability of this device is %d.%d\n",prop.major,prop.minor);
	//printf("Number of Multiprocessors = %d\n", prop.multiProcessorCount);
	//printf("Max no. of blocks allowed in a 1D Grid = %d\n", prop.maxGridSize[0]);
	//printf("Max no. of threads allowed in 1D block = %d\n", prop.maxThreadsDim[0]);
	//printf("Max no. of threads allowed in a block = %d\n", prop.maxThreadsPerBlock);
	//printf("No. of registers per block = %d\n", prop.regsPerBlock);
	//printf("Shared Memory Per block (in KB) = %f\n", (float)prop.sharedMemPerBlock * 1.0E-03);
	printf("Total Global Memory available = %f GB\n",(float)prop.totalGlobalMem * 1.0E-09);
	printf("Warp Size in threads = %d\n",prop.warpSize);
}


///////////////////////////////////////////////////////////////////////////////////////////////////////////
// Wrapper Fuctions for error checking
//////////////////////////////////////////////////////////////////////////////////////////////////////////

void cudaSafeMalloc(void ** devicePtr, size_t size)
{
	hipMalloc(devicePtr, size);
	checkCudaErrors();
}

void CudaGetDeviceProperties(hipDeviceProp_t *devicePropPtr, int deviceId)
{
	hipGetDeviceProperties(devicePropPtr, deviceId);
	checkCudaErrors();
}

void CudaGetDevice(int *deviceIdPtr)
{
	hipGetDevice(deviceIdPtr);
	checkCudaErrors();
}

void checkCudaErrors()
{
	hipError_t error = hipGetLastError();
	if(error != hipSuccess)
	{   
   	printf("Cuda Error: %s\n",hipGetErrorString(error));
   	hipDeviceReset();
		exit(-1);
	}
}
