/*******************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                        October 15-18, 2013

File          :  MemcpyLatencyHiding.cu  

Description   :  This program is designed to demonstrate some strategy to hide bandwith latency
	         and do concurrent execution of some execution kernel through one stream, while
	         memory copy of data set is also going one for the purpose of executon through
	         other kernel.

Input	      : Matrix Dimension. [ specified through preprocessor directive. ] 


Output        : This primary version of code with un-optimized kernel shows 3 to 4 percent 
		improvement in execution time because of applying ladder execution model 
		approach. 
		The Output shows the comparision of execution time of executing same set of task 
		in both mode[ no-stream and stream ].

Created       : August-2013

E-mail        : hpcfte@cdac.in     

**********************************************************************/

#include<stdio.h>
#include<hip/hip_runtime.h>
#include<math.h>
#include<time.h>
#include "ComputingKernel.cu"  

#define NSTREAM 3          // number of stream will be used for execution : this also specify, 
                           // the type of execution. currently we are  doing one three king
                           // of execution mat-mat-mult, mat-transpose- and mat-scal-mult.

#define BLOCK_SIZE 16      // Thread Block Dimension

/************************************************************************************

* Number of repetation of execution of algorithm through stream

************************************************************************************/
#define REP_COUNT 1
//#define REP_COUNT 2
//#define REP_COUNT 4
//#define REP_COUNT 8
//#define REP_COUNT 16
//#define REP_COUNT 32
//#define REP_COUNT 64


/************************************************************************************
* Define matrix dimension for execution
************************************************************************************/
//#define MAT_DIMM  1024
//#define MAT_DIMM  2048
#define MAT_DIMM  4096
//#define MAT_DIMM  8192

/************************************************************************************
* Show info regarding code
************************************************************************************/
void info()
 {
   printf("\n---------------------------------------------------------------------\n");
   printf(" Kernels : A = A + B, A = A(T) , A = x * A   \n \t\t[Where A and B is Block Matrix]");
   printf("\n Matrix Dimension : %d", MAT_DIMM);
   printf("\n Number of execution stream  : %d", NSTREAM);
   printf("\n---------------------------------------------------------------------\n");
 }

/************************************************************************************
* Memory Copy latency hide function
************************************************************************************/
int memcpyLatencyHide()
 {
     int count, rCount;
     hipStream_t *stream = (hipStream_t*) malloc ( NSTREAM * sizeof(hipStream_t));
  // define all host matrix block
     float *hTransMat, *hAddMatMatA, *hAddMatMatB, *hMatScaler;
  
  // define device matrix block
     float *dTransMat, *dTransMatOut; 
     float *dAddMatMatA, *dAddMatMatB;  
     float *dMatScaler; 	
    
  // allocate and initialize an array of stream handles
     for(count = 0; count< NSTREAM; count++)
	CUDA_SAFE_CALL( hipStreamCreate(&(stream[count])));
 
  // allocate memory at host
     CUDA_SAFE_CALL( hipHostMalloc((void**)&hTransMat , MAT_DIMM * MAT_DIMM * sizeof(float)));
     CUDA_SAFE_CALL( hipHostMalloc((void**)&hAddMatMatA , MAT_DIMM * MAT_DIMM * sizeof(float)));
     CUDA_SAFE_CALL( hipHostMalloc((void**)&hAddMatMatB , MAT_DIMM * MAT_DIMM * sizeof(float)));
     CUDA_SAFE_CALL( hipHostMalloc((void**)&hMatScaler , MAT_DIMM * MAT_DIMM * sizeof(float)));

  // assign value to input matrises
     for(count=0; count< MAT_DIMM * MAT_DIMM; count++){
	hTransMat[count] = rand() * 2.109;
	hAddMatMatA[count] = rand() * 1.02;
	hAddMatMatB[count] = rand() * 1.99;
	hMatScaler[count] = rand() * 1.11;
     }
  
  // allocate device memory
     CUDA_SAFE_CALL( hipMalloc((void**) &dTransMat, MAT_DIMM * MAT_DIMM * sizeof(float))); 
     CUDA_SAFE_CALL( hipMalloc((void**) &dTransMatOut, MAT_DIMM * MAT_DIMM * sizeof(float))); 
     CUDA_SAFE_CALL( hipMalloc((void**) &dAddMatMatA, MAT_DIMM * MAT_DIMM * sizeof(float))); 
     CUDA_SAFE_CALL( hipMalloc((void**) &dAddMatMatB, MAT_DIMM * MAT_DIMM * sizeof(float))); 
     CUDA_SAFE_CALL( hipMalloc((void**) &dMatScaler, MAT_DIMM * MAT_DIMM * sizeof(float))); 

  // define cuda event variable and create handles
     //hipEvent_t start, stop;
     //CUDA_SAFE_CALL( hipEventCreate(&start));
     //CUDA_SAFE_CALL( hipEventCreate(&stop));
 
  // define kernel dimension 
     dim3 transGrid(1,1), addGrid(1,1), scalGrid(1,1);
     dim3 transBlock(BLOCK_SIZE, BLOCK_SIZE), addBlock(BLOCK_SIZE, BLOCK_SIZE), scalBlock(BLOCK_SIZE, BLOCK_SIZE);


     timestamp(" Starting Stream execution Block :  ");
     //-------------------------------------------
 for(rCount = 0; rCount < REP_COUNT; rCount++)
   {
     CUDA_SAFE_CALL( hipMemcpyAsync(dTransMat, hTransMat, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice, stream[0])); 
     //-------------------------------------------
     MatTranspose<<<transGrid, transBlock,128, stream[0]>>>( dTransMat, dTransMatOut, MAT_DIMM,BLOCK_SIZE);
     CUDA_SAFE_CALL( hipMemcpyAsync(dAddMatMatA, hAddMatMatA, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice, stream[1])); 
     CUDA_SAFE_CALL( hipMemcpyAsync(dAddMatMatB, hAddMatMatB, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice, stream[1])); 
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpyAsync(hTransMat,dTransMatOut, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyDeviceToHost, stream[0])); 
     MatAdd<<<addGrid, addBlock, 128, stream[1]>>>(dAddMatMatA,dAddMatMatB, MAT_DIMM,BLOCK_SIZE);
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpyAsync(dMatScaler,hMatScaler, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice, stream[2])); 
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpyAsync(hAddMatMatA, dAddMatMatA, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyDeviceToHost, stream[1])); 
     scalMatMult<<<scalGrid,scalBlock,128,stream[2]>>>(dMatScaler,(float)(rand()*2.99),MAT_DIMM, MAT_DIMM, BLOCK_SIZE);
   }
     hipDeviceSynchronize();
     timestamp(" End Stream execution Block :  ");
   
   
     timestamp("\n Starting Non Stream Execution Block :  ");
 for(rCount = 0; rCount<REP_COUNT; rCount++)
  {
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpy(dTransMat, hTransMat, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice)); 
     MatTranspose<<<transGrid, transBlock,128>>>( dTransMat, dTransMatOut, MAT_DIMM,BLOCK_SIZE);
     CUDA_SAFE_CALL( hipMemcpy(dAddMatMatA, hAddMatMatA, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice)); 
     CUDA_SAFE_CALL( hipMemcpy(dAddMatMatB, hAddMatMatB, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice)); 
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpy(hTransMat,dTransMatOut, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyDeviceToHost)); 
     MatAdd<<<addGrid, addBlock, 128>>>(dAddMatMatA,dAddMatMatB, MAT_DIMM, BLOCK_SIZE);
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpy(dMatScaler,hMatScaler, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyHostToDevice)); 
     //-------------------------------------------
     CUDA_SAFE_CALL( hipMemcpy(hAddMatMatA, dAddMatMatA, MAT_DIMM * MAT_DIMM * sizeof(float), hipMemcpyDeviceToHost)); 
     scalMatMult<<<scalGrid,scalBlock,128>>>(dMatScaler,(float)(rand()*2.99),MAT_DIMM, MAT_DIMM, BLOCK_SIZE);
  }
     hipDeviceSynchronize();
     timestamp(" End Non Stream execution Block :  ");
     printf("\n---------------------------------------------------------------------\n");

 return 0; 
 }// end of strassenMatMat

int main(int argc, char* argv[])
 {
   info();
   memcpyLatencyHide();
 }// end of main 
