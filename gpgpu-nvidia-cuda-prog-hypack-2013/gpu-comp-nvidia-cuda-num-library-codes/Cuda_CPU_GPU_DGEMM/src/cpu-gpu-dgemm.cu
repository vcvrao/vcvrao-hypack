#include "hip/hip_runtime.h"
/*********************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

Objective : To carry matrix matrix multiplication by partition the Matrix B 
            between CPU+GPU using CBLAS+CUBLAS - BLAS3 library functions	 
Input	  : Size of the matrix row size, matrix column size 
Output	  : Time Taken for computation , Gflop/s 

Created   : August-2013

E-mail    : hpcfte@cdac.in     

********************************************************************************/

#include <stdio.h>
#include <stdlib.h>
# include <sys/time.h>

/*CUBLAS library initialization */
#include <hip/hip_runtime.h>
#include "hipblas.h"
#include <hip/hip_runtime.h>

/* CBLAS header file inclusion*/
extern "C" 
{
	#include <mkl.h>
//	#include <cblas.h>;
}


#define EPS 1.0e-14 /* threshhold aprrox epsilion value */ 

#define TOTALEVENT 5  /* Toltal number of GPU event to be recorded */

#define LINE_DOT "\n..................................................................................\n"
#define LINE "\n_________________________________________________________________________________________\n"

/* Function declaration of function which checks GPU results against CPU results */
void  checkResult(double *InMatA, double *InMatB, double *outMatC, int m,int n , int k);

/* Main function */
int main(int argc, char** argv)
{
	int 		rowA, rowB, rowC, colA, colB, colC; /* holds matrices dimensions */
	int		M, K, N, i, n_cpu, n_gpu, lda, ldb, ldc; /* holds matrices dimension and leading dimension */
	double		alpha = 1.0, beta = 0.0; 
	double*		hMatA;	/* host matrix A */
	double*		hMatB;  /* host matrix B */
	double*		hMatC;  /* host matrix C */
	double*		dMatA;	/* device matrix A */
	double*		dMatB;  /* device matrix B */
	double*		dMatC;  /* device output matrix C */
	float 		*elapsedTime; /* holds total elapsed time */
	double  	mflops, gflops;	/* holds FLOPS */
	double 		Tsec_cpu=0.0,Tsec_gpu=0.0; /* holds time taken for computation on CPU, GPU */
        struct 	timeval	tv_start, tv_end; /* variable to hold start / end time for CPU*/
	hipEvent_t  	*start, *stop;
	hipblasStatus_t 	status;		/* holds status of function calls */
 	hipError_t     Error;		/* holds error return from function calls */

	hipDeviceProp_t  prop; /* device property */
        int             device;	/* holds number of devices */


	//printf("\n ***************** Code  : CPU + GPU Dgemm (BLAS3 + CUBLAS3) ***********************\n");

	if (argc != 4 ){
		printf("\n Invalid number of arguments : <./executable>  <rowA > <colA/rowsB> <colB> \n");
		exit(-1);
	}	
	
	/********************************************
         Device Properties capturing
	*********************************************/
        Error = hipGetDevice (&device);
        if (Error != hipSuccess){
        printf ("%s", Error);
	exit (-1);
	}

        Error = hipGetDeviceProperties (&prop, device);
        if (Error != hipSuccess){
        printf ("%s", Error);
	exit (-1);
	}
        else{
      // printf ("\n\t Device name: %s", prop.name);
      // printf ("\n\t Majornumber: %d", prop.major);
      // printf ("\n\t Minornumber: %d", prop.minor);
        }

	/**************************************************
         Verifying Compute Capabilities >= 1.3
	***************************************************/
        if ((prop.major >= 1 && prop.minor >= 3) || (prop.major >= 2 && prop.minor >= 0)){
      // printf ("\n\t Device is capable of double precision computation !! \n");
	}
        else{
        printf ("\n\t Device is NOT capable of double precision computation !! \n Exiting ... \n");
        exit (-1);
        }

	 /* read matrices size from command line arguments */
	 M = atoi(argv[1]) ;
         K = atoi(argv[2]);
         N = atoi(argv[3]);

         
	if ( N % 2 !=0 ) {
	 	 printf("\n  Usage : <./executable> <rowA > <colA/rowsB> <colB> ");
               	 printf(" \n Column of Matix B [ argv[3] ] should be divisible by 2 .. exiting \n\n");
                exit(-1); 


	}
	
	rowA = M ; /* set rows of matrix A */
  	colA = rowB = K ; /* set column of matrix A and rows of matrix B */
        colB = N ; /* set column of matrix B*/
		
	rowC = M ; /* set rows of resultant matrix C */
	colC = N ; /* set columns of resultant matrix C */


	/* Division of Data between CPU / GPU */
	n_gpu = (N - (N/2));
        n_cpu = (N - n_gpu); 

        lda = M ; ldb = K ; ldc = M; /* set the leading dimension of matrices */


	/* allocate memory for GPU events */
	start = (hipEvent_t*) malloc (sizeof(hipEvent_t)* TOTALEVENT );
	stop = (hipEvent_t*) malloc (sizeof(hipEvent_t) * TOTALEVENT);
	elapsedTime = (float*) malloc (sizeof(float) * TOTALEVENT);
	

	/* CUBlas in itialization */
	status = cublasInit();
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS initialization error !!\n");
		exit (0);
	}

 	/* Memory Allocation of the Matrices on the Host using Pinned Memory */
        Error = hipHostMalloc ((void **)&hMatA, sizeof(double) * rowA * colA);
        if (Error != hipSuccess){
        printf ("\n Error in Host Memory Allocation - Matrix A \n");
        exit (-1);
        }

        Error = hipHostMalloc ((void **)&hMatB, sizeof(double) * rowB * colB);
        if (Error != hipSuccess){
        printf ("\n Error in Host Memory Allocation - Matrix B \n");
        exit (-1);
        }

        Error = hipHostMalloc ((void **)&hMatC, sizeof(double) * rowC * colC);
        if (Error != hipSuccess){
        printf ("\n Error in Host Memory Allocation - Matrix C \n");
        exit (-1);
	}

	/*************************************************************************
         * Un-comment the below section for memory allocation using Pageable Memory
	*************************************************************************/

/*

 	// Memory Allocation of the Matrices on the Host using pageable  Memory 
 	 hMatA = (double*) malloc (rowA * colA * sizeof(double));
        if (hMatA == 0){
        printf("\n Memory allocation Failed for Matrix A");
        exit (0);
        }

        hMatB = (double*) malloc (rowB * colB * sizeof(double));
        if (hMatB == 0){
        printf("\n Memory allocation Failed for Matrix B");
        exit (0);
        }

        hMatC = (double*) malloc (rowC * colC * sizeof(double));
        if (hMatC == 0){
        printf("\n Memory allocation Failed for Resultant Matrix");
        exit (0);
        }

*/
	/* Filling the Matrix with Values*/
	for (i = 0; i < rowA * colA; i++){
		hMatA[i] = drand48();
	}
	for (i=0; i < rowB * colB; i++){
		hMatB[i] = drand48(); 
	}
	for (i=0; i < rowC * colC; i++){
		hMatC[i] = 0.0 ; 
	}

 
	/* Allocating memory for Matrices on Device*/
	status = cublasAlloc (rowA * colA, sizeof(double), (void**)&dMatA);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS Memory Allocation error for dMatA!! \n");
		exit (0);
	}
	
	status = cublasAlloc (rowB * n_gpu, sizeof(double), (void**)&dMatB);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS Memory Allocation error dMatB!! \n");
		exit (0);
	}
	status = cublasAlloc (rowC * n_gpu , sizeof(double), (void**)&dMatC);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS Memory Allocation error for dMatC!! \n");
		exit (0);
	}

	for ( i = 0; i<TOTALEVENT ; i++) {
	
		/* Creating the Events */
        	hipEventCreate (&start[i]);
        	hipEventCreate (&stop[i]);
	}
	
	hipEventRecord (start[0], 0);
	/* Initialization of device matrix A with host matrix A data */
	status = hipblasSetMatrix (rowA, colA, sizeof(double), hMatA, lda , dMatA, lda);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS Value Initialization error of dMatA!! \n");
		exit (0);
	}
	hipEventRecord (stop[0], 0);
  	hipEventSynchronize (stop[0]);	
	
	hipEventRecord (start[1], 0);
	/* Initialization of device matrix B  with host matrix B data */
	status = hipblasSetMatrix (rowB, n_gpu, sizeof(double),hMatB, ldb, dMatB, ldb);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS Value Initialization error of matrix dMatB!! \n");
		exit (0);
	}
	hipEventRecord (stop[1], 0);
  	hipEventSynchronize (stop[1]);	


	hipEventRecord (start[2], 0);
	status = hipblasSetMatrix (rowC, n_gpu, sizeof(double),hMatC, ldc, dMatC, ldc);
	if (status != HIPBLAS_STATUS_SUCCESS){
		printf("\n CUBLAS Value Initialization error of matrix dMatC!! \n");
		exit (0);
	}
	hipEventRecord (stop[2], 0);
  	hipEventSynchronize (stop[2]);	

	hipEventRecord (start[3], 0);
	/*** Performs operation using CUBLAS DGEMM ******/
	hipblasDgemm ('N', 'N', M , n_gpu , K  , alpha, dMatA, lda, dMatB, ldb , beta, dMatC, ldc );
	hipEventRecord (stop[3], 0);
  	hipEventSynchronize (stop[3]);	

	
	gettimeofday(&tv_start,0);
	/***** Performs operation using CBLAS DGEMM*******/
	cblas_dgemm (CblasColMajor, CblasNoTrans, CblasNoTrans, M, n_cpu , K, alpha, hMatA, lda , hMatB+K*n_gpu , ldb, beta, hMatC+M*n_gpu, ldc);
	gettimeofday(&tv_end,0);

	
	 hipEventRecord (start[4], 0);	
	/**** copy output matrix from device to host */
	status = hipblasGetMatrix (M, n_gpu, sizeof(double), dMatC, ldc, hMatC, ldc);
        if (status != HIPBLAS_STATUS_SUCCESS){
        printf("\n CUBLAS Error in reading the result back !! \n");
        exit (0);
	}
	 hipEventRecord (stop[4], 0);
  	hipEventSynchronize (stop[4]);	
	

	/* compute elapsed time for each operation */
	for ( i=0 ; i< TOTALEVENT ; i++) {
		hipEventElapsedTime ( &elapsedTime[i], start[i], stop[i]);
	}

	/* compute total time taken for computation on CPU */
	Tsec_cpu = ((double)tv_end.tv_sec + (((double) (tv_end.tv_usec) /1000000.0))) - ((double)tv_start.tv_sec + (((double) (tv_start.tv_usec) /1000000.0)))  ;
	
	/* Compute total computation on GPU */
	for ( i=0 ; i< TOTALEVENT ; i++) {
		Tsec_gpu += (double) (elapsedTime[i]  * 1.0e-3);
	}


	/* compute FLOPS */
	mflops= ((1.0e-6 * ((2. * M *K * n_cpu) /Tsec_cpu)) + (1.0e-6 * (( 2.0 * M * K * n_gpu)/Tsec_gpu)));
	gflops= (1.0e-3 * mflops );


        printf(LINE_DOT);
        printf("Matrix-Size \t Comp Time(CPU+GPU Sec) \t  CBLAS GFlops ");
        printf(LINE_DOT);
        printf("\n %d * %d  \t %.8lf  \t\t\t %.8lf", M, N,(Tsec_cpu + Tsec_gpu)  , gflops );
        printf(LINE_DOT);


	/* check CPU+GPU results against CPU results */
        checkResult( hMatA, hMatB, hMatC,M,N,K);
        printf(LINE);
        printf("\n");

/*
	printf("\n Printing resultant Matrix \n");
	for (i=0; i < M * N; i++){
		printf("\t %f \n",hMatC[i]);
	}

*/


	/* Destroy cuda event */
	 for ( i=0 ; i< TOTALEVENT ; i++) {
                hipEventDestroy(start[i]);
                hipEventDestroy(stop[i]);
        }




	 /* Free the memory on Host */
        Error = hipHostFree(hMatA);
        if (Error != hipSuccess){
        printf("\n Error in freeing the memory of matrix A on Host !! \n ");
        exit (0);
        }

        Error = hipHostFree(hMatB);
        if (Error != hipSuccess){
        printf("\n Error in freeing the memory of matrix B on Host !! \n");
        exit (0);
        }

        Error = hipHostFree(hMatC);
        if (Error != hipSuccess){
        printf("\n Error in freeing the memory of matrix C on Host !! \n");
        exit (0); 
        }


	/* Free the memory on Device */
	status = cublasFree(dMatA);
	if (status != HIPBLAS_STATUS_SUCCESS){
	printf("\n CUBLAS Error in freeing the memory of matrix A on Device !! \n");
	exit (0);
	}
	
	status = cublasFree(dMatB);
	if (status != HIPBLAS_STATUS_SUCCESS){
	printf("\n CUBLAS Error in freeing the memory of matrix B on Device !! \n");
	exit (0);
	}
	
	status = cublasFree(dMatC);
	if (status != HIPBLAS_STATUS_SUCCESS){
	printf("\n CUBLAS Error in freeing the memory of matrix C on Device !! \n");
	exit (0);
	}


	/* Shutdown */
	status = cublasShutdown();
	if (status != HIPBLAS_STATUS_SUCCESS){
	printf("\n CUBLAS Shutdwon Error !! \n");
	exit (0);
	}

	free(start);
        free(stop);
        free(elapsedTime);

	 /********************************
         Un-comment the below section if memory is allocated using pageable memory
        **********************************/


/*   
	 free(hMatA);
        free(hMatB);
        free(hMatC);

*/

}

/***********************************************************************************
Routine for verifiying the CPU+GPU results against the CPU results
************************************************************************************/
void checkResult(double *InMatA, double *InMatB, double *outMatC, int m, int n , int k )
{
        double  alpha = 1.0, beta = 0.0;
        int     i;
        //int     j;
        //int     k1;
        double  *tempOut;
        double  errorNorm = 0.0;
        double  eps=EPS;
        double  relativeError=0.0;
	int 	flag=0;

        tempOut  = (double*) malloc (m * n * sizeof(double));
        if (tempOut == 0){
        printf("\n Memory allocation Failed for Resultant Matrix");
        exit (0);
        }

	/* CPU Compuation Performs operation using CBLAS */
        cblas_dgemm (CblasColMajor, CblasNoTrans, CblasNoTrans, m, n , k, alpha, InMatA, m , InMatB , k, beta, tempOut, m);

        /****************************************************************** 
	Serial computation
	uncomment the below section if want to do the CPU computation
	using i,j,k loop method. Method work only for square matrices.
	 *******************************************************************/
       /*  for (i = 0; i < n  ; ++i) {
                for (j = 0; j < n; ++j) {
                        double  cprod = 0;
                        for (k1 = 0; k1 < n; ++k1) {
                                cprod += InMatA[k1 * n + i] * InMatB[j * n + k1];
                        }
                tempOut[j * n + i] = alpha * cprod + beta * tempOut[j * n + i];

                }
        }
	*/

        /*** check relative error with approx precision ****/
        for( i = 0; i < m*n; ++i) {

                if (fabs(tempOut[i]) > fabs(outMatC[i]))
                        relativeError = fabs((tempOut[i] - outMatC[i]) / tempOut[i]);
                else
                        relativeError = fabs((outMatC[i] - tempOut[i]) / outMatC[i]);

                if (relativeError > eps && relativeError != 0.0e+00 ){
                        if(errorNorm < relativeError) {
                        errorNorm = relativeError;
			flag=1;
                        }
                }

        }
	if( flag == 1) {

		//printf(" \n Results verfication : Failed");
		printf(" \n Considered machine precision : %e", eps);
		printf(" \n Relative Error 		    : %e", errorNorm);	

	}
	else {

		printf("\n Results verfication : Success");
	}

	free(tempOut);
}

