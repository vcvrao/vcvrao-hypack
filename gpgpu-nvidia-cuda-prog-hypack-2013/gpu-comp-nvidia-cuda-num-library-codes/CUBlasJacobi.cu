/****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Objective   : program to solve a  linear system of equations (Ax = b) using jacobi 
                method in a GPU (using cublas lib)

  Input       : Number of unknowns and maximum number of iterations

  Output      : Solution Vector.

  Created     : August-2013

  E-mail      : hpcfte@cdac.in     

****************************************************************************/

#include<stdio.h>
#include<stdlib.h>
#include<math.h>
#include<sys/time.h>
#include<hip/hip_runtime.h>
#include "hipblas.h"
//-------------------------------------------------------------------------------------------------------------------------
#define BLOCKSIZE 16 
#define ERR 0
#define DOMINANCE 1
//-------------------------------------------------------------------------------------------------------------------------
void   InitializeVectors(float** coeffMat, float** solutionVect, float** rhsVect, float** diagOfCoeffMat, float** tempSolution, int Size, double *diag);
void   generateCoeffMat(int Size, float** coeffMat, double *diag);
void   generateRhsVect(int Size, float** rhsVect, float* coeffMat);
void   generateSolutionVect(int Size, float** solutionVect);
void   generateDiagOfCoeffMat(int Size, float** diagOfCoeffMat, float* coeffMat);
void   generateTempSolution(float** tempSolutioni,int);
float  calcDiff(float* solutionVect, float* tempSolution, int Size);
void   outputSolution(float* solutionVect, int iteration, double timing, int Size);
void   freeHostMemory(float* coeffMat, float* solutionVect, float* rhsVect, float* diagOfCoeffMat, float* tempSolution);
void*  malloc_safe_call(int size);
//-------------------------------------------------------------------------------------------------------------------------

/************************************************************************************
*  pragma routine to report the detail of cuda error
************************************************************************************/
#define CUDA_SAFE_CALL(call)                                                    \
            do{                                                                 \
                hipError_t err = call;                                         \
                if(err != hipSuccess)                                          \
                 {                                                              \
                   fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                   __FILE__, __LINE__, hipGetErrorString( err) );              \
                   exit(1);                                                     \
                 }                                                              \
             } while (0)                                                        \

//---------------------------------------------------------------------------------------------------------------------------
//kernel that performs parallel division of scalar pairs which are given as corresponding components of two vectors.
__global__ void ParallelScalarDiv( float* vector1, float* vector2, float *result, int vectDim, int threadDim)
{
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;
	int tindex = (threadDim * tidx) + tidy;
    	int maxNumThread = threadDim * threadDim;
	int pass = 0;  
	int threadColumnIdx;
	while( (threadColumnIdx = (tindex + maxNumThread * pass))  < vectDim )
	{
		result[threadColumnIdx] = vector1[threadColumnIdx] / vector2[threadColumnIdx];            
		pass++;
	}
	__syncthreads();
}
//---------------------------------------------------------------------------------------------------------------------------

int main(int argc, char *argv[])
{	
	//checking if valid number of arguments have been passed
	if(argc != 3)
	{
		printf("usage -> <./programName> <Number of unknowns> <maximum iterations>\n");
		exit(-1);
	}
	
	int Size = atoi(argv[1]);	//obtaining dimension of co-eff matrix from the argument passed by user
	int maxItr = atoi(argv[2]);	//the maximum number of iterations the program is allowed to run
	float  diffX = 0;	 	//difference between two result vectors of successive iterations
	double diag;			//diagonal element of coefficient matrix
	int iteration = 0; 	 	//iteration count
	int i;
	struct timeval tv;
	double timing;			//time taken for execution
	
	//host vectors
	float* coeffMat;         	//coeff matrix	
	float* rhsVect;          	//RHS vector
	float* solutionVect;    	//result vector
	float* diagOfCoeffMat;   	//diagonal elements of coeffMat
	float* tempSolution;	 	//new values calculated for result vector in the subsequent iteration
	
	//device vectors
	float* devCoeffMat;
	float* devRhsVect;
	float* devSolutionVect;
	float* devDiagOfCoeffMat;
	float* devTempResult; 
			               
	//generating and initalizing the required vectors in the host
	InitializeVectors(&coeffMat, &solutionVect, &rhsVect, &diagOfCoeffMat, &tempSolution, Size, &diag);
	
	//allocation of device memory
  	CUDA_SAFE_CALL(hipMalloc( (void**)&devSolutionVect, Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devCoeffMat, Size * Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devRhsVect, Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devDiagOfCoeffMat, Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devTempResult, Size * sizeof(float)));

	//copying host to device 
	CUDA_SAFE_CALL(hipMemcpy((void*)devCoeffMat, (void*)coeffMat, Size*Size*sizeof(float) , hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(hipMemcpy((void*)devRhsVect, (void*)rhsVect, Size*sizeof(float) , hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(hipMemcpy((void*)devSolutionVect, (void*)solutionVect, Size*sizeof(float) , hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(hipMemcpy((void*)devDiagOfCoeffMat, (void*)diagOfCoeffMat, Size*sizeof(float) , hipMemcpyHostToDevice ));
	
	//defining thread grid and block
	dim3 dimGrid(1,1);
	dim3 dimBlock(BLOCKSIZE, BLOCKSIZE); 
	
	//intializing cublas
	cublasInit();
	
	//recording start time
	gettimeofday(&tv, NULL);
   	double t1=tv.tv_sec+(tv.tv_usec/1000000.0);
	
	//performing jacobian method to obtain solution parallely by calling the device kernels
   	do
	{
		//incrementing the iteration count
		iteration++;
				
		//multiplying coeffMat and solutionVect and subtracting from RHS vector (b-Ax)
		cublasScopy(Size, devRhsVect, 1, devTempResult, 1);

	        cublasSgemv ('N', Size, Size, -1, devCoeffMat, Size, devSolutionVect, 1, 1, devTempResult, 1);

		cublasScopy(Size, devTempResult, 1, devSolutionVect, 1);

		//dividing the result of the vector-vector subtraction by the diagonal elements
		ParallelScalarDiv<<<dimGrid, dimBlock>>>( devTempResult, devDiagOfCoeffMat, devSolutionVect, Size, BLOCKSIZE);
		
		//copying the result from the device
		CUDA_SAFE_CALL(hipMemcpy((void*)tempSolution, (void*)devSolutionVect, Size * sizeof(float), hipMemcpyDeviceToHost ));		
		
		//calculating difference in solutionVect in successive iterations (eucledian distance)
		diffX = calcDiff(solutionVect, tempSolution, Size);
		
		//assigning tempSolution to solutionVect
		for(i=0; i<Size; i++)
		{
			solutionVect[i] = tempSolution[i];
		}

	}while(diffX > ERR && iteration < maxItr);

	//recording stop time
	gettimeofday(&tv,NULL);
   	double t2=tv.tv_sec+(tv.tv_usec/1000000.0);
	
	//calculating time taken for computation
	timing = t2 - t1;	

	cublasShutdown();
	
 	//outputting the solution to both stdout and a file
	outputSolution(solutionVect, iteration, timing, Size);
	
	//freeing the memory allocated to the diffrerent vectors on host
	freeHostMemory(coeffMat, solutionVect, rhsVect, diagOfCoeffMat, tempSolution);
	
	//freeing the memory allocated to the diffrerent vectors on device
	hipFree(devSolutionVect);
	hipFree(devCoeffMat);
	hipFree(devRhsVect);
	hipFree(devDiagOfCoeffMat);
	hipFree(devTempResult);
	
	return 0;	
}
//---------------------------------------------------------------------------------------------------------------------------
void InitializeVectors(float** coeffMat, float** solutionVect, float** rhsVect, float** diagOfCoeffMat, float** tempSolution, int Size, double *diag)
{
	generateCoeffMat(Size, coeffMat, diag);
	generateRhsVect(Size, rhsVect, *coeffMat);
	generateSolutionVect(Size, solutionVect);
	generateDiagOfCoeffMat(Size, diagOfCoeffMat, *coeffMat);
	generateTempSolution(tempSolution,Size);
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateCoeffMat(int Size, float** coeffMat, double *diag)
{
	int i,j;
	*diag = 0;  //sum of the row element of the coeff matrix
	
	//assigining and verifying availability of memory	
	*coeffMat = (float *)malloc_safe_call(Size*Size*sizeof(float));	
	
	//assigning values to the coefficient matrix in column major order
	for(j=0; j<Size; j++)
		for(i=0; i<Size; i++)
			(*coeffMat)[i+j*Size] = j+1;

	//finding sum of row elements and making it diagonal element
	for(i=0; i<Size; i++)
		(*diag) += (*coeffMat)[i*Size];

	(*diag) = (*diag) * DOMINANCE;

	//making the matrix diagonal dominant
	for(i=0; i<Size; i++)
		(*coeffMat)[i*Size + i] = (*diag);
	return ;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateRhsVect(int Size, float** rhsVect, float* coeffMat)
{
	int i,j;
		
	//assigining and verifying availability of memory	
	*rhsVect = (float *)malloc_safe_call(Size*sizeof(float));
	
	//initializing rhsVect to 0's
	for(i=0; i<Size; i++)
		(*rhsVect)[i] = 0;
	//assigning values to rhsVect 
	for(i=0; i<Size; i++)
		for(j=0; j<Size; j++)
			(*rhsVect)[i] += coeffMat[j*Size + i];
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateSolutionVect(int Size, float** solutionVect)
{
	int i;
	
	//assigining and verifying availability of memory
	*solutionVect = (float *)malloc_safe_call(Size*sizeof(float));
	
	//initial solution for x vector
	for(i=0; i<Size; i++)
		(*solutionVect)[i] = 0;
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateDiagOfCoeffMat(int Size, float** diagOfCoeffMat, float* coeffMat)
{
	int i=0;
	
	//assigining and verifying availability of memory
	*diagOfCoeffMat = (float *)malloc_safe_call(Size*sizeof(float));
	
	//values of diagOfCoeffMat
	for(i=0; i<Size; i++)
		(*diagOfCoeffMat)[i] = coeffMat[i*Size+i];
	//forcing diag elements of coeffMat matrix to zero for purpose of calculation
	for(i=0; i<Size; i++)
		coeffMat[i*Size+i] = 0;
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateTempSolution(float** tempSolution,int Size)
{
	//assigining and verifying availability of memory
	*tempSolution = (float *)malloc_safe_call(Size*sizeof(float));
	
	return;
}
//--------------------------------------------------------------------------------------------------------------------------
float  calcDiff(float* solutionVect, float* tempSolution, int Size)
{
	int i=0;
	float diffX = 0;
	for(i=0; i<Size; i++)
		diffX += pow((tempSolution[i] - solutionVect[i]), 2);
	diffX = sqrt(diffX);
	return(diffX);
}
//--------------------------------------------------------------------------------------------------------------------------
void   outputSolution(float* solutionVect, int iteration, double timing, int Size)
{
	int i=0;
	FILE *fp;	//file pointer
	
	//opening file to write solution
	fp = fopen("gpuCublasSolution.txt","w+");
	if(fp == NULL)
	{
		printf("Can't open the file gpuCublasSolution\n");
		exit(-1);
	}
	
	//displaying final result
	for(i=0; i<Size; i++)
	{
		printf("%f\n", solutionVect[i]);	//printing to stdout
		fprintf(fp, "%f\n", solutionVect[i]);   //printing to file
	}	
	
	printf("\nx vector displayed above is calculated in %d iterations and in %lf secs\n\n", iteration, timing);
	fclose(fp);
		
}
//--------------------------------------------------------------------------------------------------------------------------
void   freeHostMemory(float* coeffMat, float* solutionVect, float* rhsVect, float* diagOfCoeffMat, float* tempSolution)
{
	free(coeffMat);
	free(solutionVect);
	free(rhsVect);
	free(diagOfCoeffMat);
	free(tempSolution);
	
	return;
}
//--------------------------------------------------------------------------------------------------------------------------
void *malloc_safe_call(int size)
{
	void *ptr;
	
	ptr = malloc(size);
	
	if(ptr==NULL)
	{
		printf("memory unavailable\n");
		exit(-1);
	}	
	
	return(ptr);	
}
//--------------------------------------------------------------------------------------------------------------------------
