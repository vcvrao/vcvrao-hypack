/***************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

  Example     : cuda-matrix-matrix-addition.cu
 
  Objective   : Write CUDA program to compute Matrix-Matrix addition.

  Input       : None 

  Output      : Execution time in seconds , Gflops achieved
                                                                                                                            
  Created     : August-2013

  E-mail      : hpcfte@cdac.in     

****************************************************************************/

#include<stdio.h>
#include<hip/hip_runtime.h>

#define EPS 1.0e-12
#define GRIDSIZE 10
#define BLOCKSIZE 16

#define SIZE 128

int size = SIZE;
hipDeviceProp_t deviceProp;	
hipEvent_t start,stop;
hipError_t ret;

/* kernel funtion */
__global__ void add_matrix (double *matA,double *matB,double *matC,int length)
{
	int i=blockIdx.x * blockDim.x + threadIdx.x;
	int j=blockIdx.y * blockDim.y + threadIdx.y;
	int k = i+j*length;
	
	if(i<length&&j<length)
	matC[k] = matA[k]+matB[k];
	__syncthreads();
	
}

/* Check for safe return of all calls to the device */
void CUDA_SAFE_CALL(hipError_t call)
{
        hipError_t ret = call;
        //printf("RETURN FROM THE CUDA CALL:%d\t:",ret);
        switch(ret)
        {
                case hipSuccess:
                //              printf("Success\n");
                                break;
        /*      case cudaErrorInvalidValue:
                                {
                                printf("ERROR: InvalidValue:%i.\n",__LINE__);
                                exit(-1);
                                break;
                                }
                case cudaErrorInvalidDevicePointer:
                                {
                                printf("ERROR:Invalid Device pointeri:%i.\n",__LINE__);
                                exit(-1);
                                break;
                                }
                case cudaErrorInvalidMemcpyDirection:
                                {
                                printf("ERROR:Invalid memcpy direction:%i.\n",__LINE__);
                                exit(-1);
                                break;
                                }                       */
                default:
                        {
                                printf(" ERROR at line :%i.%d' ' %s\n",__LINE__,ret,hipGetErrorString(ret));
                                exit(-1);
                                break;
                        }
        }
}



/* Get the number of GPU devices present on the host */
int get_DeviceCount()
{
	int count;
	hipGetDeviceCount(&count);	
	return count;	
}

/* Fill in the vector with double precision values */
void fill_dp_vector(double* vec,int size)
{
	int ind;
	for(ind=0;ind<size;ind++)
		vec[ind]=drand48();	
}


/* Function to check grid and block dimensions */
void check_block_grid_dim(hipDeviceProp_t devProp,dim3 blockDim,dim3 gridDim)
{
	
	if( blockDim.x >= devProp.maxThreadsDim[0] || blockDim.y >= devProp.maxThreadsDim[1] || blockDim.z >= devProp.maxThreadsDim[2] )
	{
		printf("\nBlock Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2]);
	       exit(-1);	
	}	
	
	if( gridDim.x >= devProp.maxGridSize[0] || gridDim.y >= devProp.maxGridSize[1] || gridDim.z >= devProp.maxGridSize[2] )
	{
		printf("\nGrid Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
	       exit(-1);	
	}	
}

/* Function to print memory error */
void mem_error(char *arrayname, char *benchmark, int len, char *type)
{

	printf("\nMemory not sufficient to allocate for array %s\n\tBenchmark : %s  \n\tMemory requested = %d number of %s elements\n",arrayname, benchmark, len, type);
	printf("\tAborting\n");
	exit(-1);
}

/* launch kernel function is called in main() */
void launch_kernel_MatMatAdd(double *device_MatA,double *device_MatB,double *device_MatC,int size)
{

	dim3 dimBlock(BLOCKSIZE,BLOCKSIZE);
	dim3 dimGrid(size/dimBlock.x,size/dimBlock.y);	
 
        /* checking the maximum limit of blocksize and gridsize */
	check_block_grid_dim(deviceProp,dimBlock,dimGrid);
  
	add_matrix<<<dimGrid,dimBlock>>>(device_MatA,device_MatB,device_MatC,size);
	
}

/* Function to calculate gflops */
double calculate_gflops(double &Tsec)
{
	//printf("time taken is %.8lf\n",Tsec);
	double gflops=(1.0e-9 * (( 1.0 * size*size )/Tsec));
	//printf("Gflops is \t%f\n",gflops);
	return gflops;

}

/* prints the result on screen */
void print_on_screen(char * program_name,float tsec,double gflops,int size,int flag)//flag=1 if gflops has been calculated else flag =0
{
	printf("\n---------------%s----------------\n",program_name);
	printf("\tSIZE\t TIME_SEC\t Gflops\n");
	if(flag==1)
	printf("\t%d\t%f\t%lf\t",size,tsec,gflops);
	else
	printf("\t%d\t%lf\t%lf\t",size,"---","---");

}

/* Function to perform Mat Addition on CPU */
void CPU_MatMatAdd(double *A,double *B,double *C,int length)
{
	for(int i =0;i<length*length;i++)
		C[i] = A[i]+B[i];
}

/* Function to check cpu and gpu results */
void relError(double* dRes,double* hRes,int size)
{
        double relativeError=0.0,errorNorm=0.0;
	int flag=0;
	int i;

	for( i = 0; i < size; ++i) {
                if (fabs(hRes[i]) > fabs(dRes[i]))
                        relativeError = fabs((hRes[i] - dRes[i]) / hRes[i]);
                else
                        relativeError = fabs((dRes[i] - hRes[i]) / dRes[i]);

                if (relativeError > EPS && relativeError != 0.0e+00 )
		{
                        if(errorNorm < relativeError) 
			{
                        	errorNorm = relativeError;
                        	flag=1;
                        }
                }

        }
        if( flag == 1) 
	{
                printf(" \n Results verfication : Failed");
                printf(" \n Considered machine precision : %e", EPS);
                printf(" \n Relative Error                  : %e\n", errorNorm);

        }
        else 
                printf("\n Results verfication : Success\n");

}

/* free memory */
void dfree(double * arr[],int len)
{
        for(int i=0;i<len;i++)
                CUDA_SAFE_CALL(hipFree(arr[i]));
        printf("mem freed\n");
}





/* main()   */

int main()
 {
	double *host_MatA,*host_MatB,*host_MatC,*CPU_Result;
	double *device_MatA,*device_MatB,*device_MatC;

	int device_Count=get_DeviceCount();
        printf("\n\nNUmber of Devices : %d\n\n", device_Count);

        /* Device Selection, Device 1 */
        hipSetDevice(0);
	
	int device;
        /* Current Device Detection */
        hipGetDevice(&device);         
        hipGetDeviceProperties(&deviceProp,device);

	printf("Using device %d: %s \n", device, deviceProp.name);

	/* event creation */
	CUDA_SAFE_CALL(hipEventCreate (&start));
        CUDA_SAFE_CALL(hipEventCreate (&stop));
   

   
       /* allocating the memory for each matrix */
	host_MatA = new double[size*size];
	host_MatB = new double[size*size];
	host_MatC = new double[size*size];
	CPU_Result= new double[size*size];
	 if(host_MatA==NULL)
                mem_error("host_MatA","matmatadd",size,"double");

	if(host_MatB==NULL)
                mem_error("host_MatB","matmatadd",size,"double");
	if(host_MatC==NULL)
                mem_error("host_MatC","matmatadd",size,"double");
	if(CPU_Result==NULL)
                mem_error("CPU_Result","matmatadd",size,"double");


	/* filling the matrix with double precisio */
  	fill_dp_vector(host_MatA,size*size);
  	fill_dp_vector(host_MatB,size*size); 

	/* filling host_MatC with 0.0 value */
	for(int i =0;i<size*size ;i++)
	host_MatC[i]=0.0;
  
 	/* allocating memory on GPU */
	CUDA_SAFE_CALL(hipMalloc( (void**)&device_MatA,size*size*sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&device_MatB, size*size*sizeof(double)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&device_MatC,size*size*sizeof(double)));

 	/* copying host matrix to device matrix */
    	CUDA_SAFE_CALL(hipMemcpy((void*)device_MatA, (void*)host_MatA, size*size* sizeof(double) , hipMemcpyHostToDevice ));
    	CUDA_SAFE_CALL(hipMemcpy((void*)device_MatB, (void*)host_MatB, size*size*sizeof(double) , hipMemcpyHostToDevice ));
    	CUDA_SAFE_CALL(hipMemcpy((void*)device_MatC, (void*)host_MatC, size*size*sizeof(double) , hipMemcpyHostToDevice ));
  
	CUDA_SAFE_CALL(hipEventRecord (start, 0));
	launch_kernel_MatMatAdd(device_MatA,device_MatB,device_MatC,size);               //launching the kernel
	CUDA_SAFE_CALL(hipEventRecord (stop, 0));
	CUDA_SAFE_CALL(hipEventSynchronize (stop));

	/* computing elapsed time */
	float elapsedTime;	
	CUDA_SAFE_CALL(hipEventElapsedTime ( &elapsedTime, start, stop));
	double Tsec = elapsedTime *1.0e-3;

	/* calling funtion for measuring Gflops */
	calculate_gflops(Tsec);
	
        /* printing the result on screen */
        print_on_screen("MAT MAT ADDITION",Tsec,calculate_gflops(Tsec),size,1);
       
   	/* retriving result from device */
      CUDA_SAFE_CALL(hipMemcpy((void*)host_MatC, (void*)device_MatC, size*size*sizeof(double) , hipMemcpyDeviceToHost ));

  	/* to get the result uncomment this part
   printf("\n ----------------------------------------------------------------------");	
	for(int i =0;i<size*size;i++)
	   printf("%lf", host_MatC[i]);*/

	/* doing computation from CPU */
	CPU_MatMatAdd(host_MatA,host_MatB,CPU_Result,size);

	/* comparing result of CPU-GPU */
	relError(CPU_Result,host_MatC,size*size);


   
	/* free the device memory */
	double *array[3];
	array[0]=device_MatA;
	array[1]=device_MatB;
	array[2]=device_MatC;
	
	dfree(array,3);
	
	/* free host memory */

	   free(host_MatA);
	   free(host_MatB);
	   free(host_MatC);
	   free(CPU_Result);

	   hipDeviceReset();

 }// end of main
