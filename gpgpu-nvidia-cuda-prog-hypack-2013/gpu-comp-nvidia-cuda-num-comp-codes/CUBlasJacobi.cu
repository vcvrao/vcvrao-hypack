#include "hip/hip_runtime.h"


/************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

Objective : Progra to solve a linear system of matrix equations (Ax = b) 
            using jacobi ethod in a GPU library functions	 

Input	  : Nuber of unknowns and maximum number of iterations 

Output	  : Solution Vector 


Created   : August-2013

E-mail    : hpcfte@cdac.in     

**************************************************************************/
                               
#include<stdio.h>
#include<stdlib.h>
#include<ath.h>
#include<hip/hip_runtime.h>
#include<sys/tie.h>
#include<unistd.h>
//--------------------------------------------------------------------------------------------------------

#define BLOCKSIZE 16 
#define ERR 0
#define DOMINANCE 1
//-----------------------------------------------------------------------------------------------------
void   InitializeVectors(float** coeffMat, float** solutionVect, float** rhsVect, float** diagOfCoeffMat, float** tepSolution, int Size);
void   generateCoeffMat(int Size, float** coeffMat);
void   generateRhsVect(int Size, float** rhsVect, float* coeffMat);
void   generateSolutionVect(int Size, float** solutionVect);
void   generateDiagOfCoeffMat(int Size, float** diagOfCoeffMat, float* coeffMat);
void   generateTepSolution(int Size, float** tempSolution);
float  calcDiff(float* solutionVect, float* tepSolution, int Size);
void   outputSolution(float* solutionVect, int iteration, double tiing, int Size);
void   freeHostMeory(float* coeffMat, float* solutionVect, float* rhsVect, float* diagOfCoeffMat, float* tempSolution);
void  *alloc_safe_call(int size);
//------------------------------------------------------------------------------------------------------------

/************************************************************************************
*  praga routine to report the detail of cuda error
************************************************************************************/
#define CUDA_SAFE_CALL(call)                                                    \
            do{                                                                 \
                hipError_t err = call;                                         \
                if(err != hipSuccess)                                          \
                 {                                                              \
                   fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",\
                   __FILE__, __LINE__, hipGetErrorString( err) );              \
                   exit(1);                                                     \
                 }                                                              \
             } while (0)                                                        \



//kernel that perfors matrix vector multiplication in the GPU
__global__ void MatrixVectMult(float* atrix, float* vector, float* result, int nrows, int ncols, int threadDim)
{
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;
	int tindex = (threadDi * tidx) + tidy;
    	int axNumThread = threadDim * threadDim;
	int vectLen = ncols;
	int pass = 0;  
	int count ;
    	int threadColunIdx;
	float tep = 0.0f;
	while( (threadColunIdx = (tindex + maxNumThread * pass))  < ncols )
	{
		tep = 0.0f;
		for( count = 0; count < vectLen; count++)
		{
	        	tep +=  matrix[threadColumnIdx * ncols + count] * vector[count];
		}	
	        result[threadColunIdx] = temp;
		pass++;
	}
	__syncthreads();
}
//--------------------------------------------------------------------------------------------------------------------------

//kernel that adds/subtracts two vectors in the GPU
__global__ void VectorVectorAddSub(float* vector1, float* vector2, int alpha, float* result, int vectDi, int threadDim)
{
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;
	int tindex = (threadDi * tidx) + tidy;
    	int axNumThread = threadDim * threadDim;
	int pass = 0;  
	int threadColunIdx;
    	while( (threadColunIdx = (tindex + maxNumThread * pass))  < vectDim )
	{
	        result[threadColunIdx] = vector1[threadColumnIdx] + alpha * vector2[threadColumnIdx];
            pass++;
	}
	__syncthreads();
}
//--------------------------------------------------------------------------------------------------------------------------

//kernel that perfors parallel division of scalar pairs which are given as corresponding components of two vectors.
__global__ void ParallelScalarDiv( float* vector1, float* vector2, float *result, int vectDi, int threadDim)
{
	int tidx = threadIdx.x;
	int tidy = threadIdx.y;
	int tindex = (threadDi * tidx) + tidy;
    	int axNumThread = threadDim * threadDim;
	int pass = 0;  
	int threadColunIdx;
	while( (threadColunIdx = (tindex + maxNumThread * pass))  < vectDim )
	{
		result[threadColunIdx] = vector1[threadColumnIdx] / vector2[threadColumnIdx];            
		pass++;
	}
	__syncthreads();
}
//---------------------------------------------------------------------------------------------------------------------------

int ain(int argc, char *argv[])
{	
	//checking if valid nuber of arguments have been passed
	if(argc != 3)
	{
		printf("Usage: <./prograName> <Number of unknowns> <maximum iterations>\n");
		exit(-1);
	}
	
	int Size = atoi(argv[1]);	//obtaining diension of co-eff matrix from the argument passed by user
	int axItr = atoi(argv[2]);	//the maximum number of iterations the program is allowed to run
	float  diffX = 0;	 	//difference between two solution vectors of successive iterations
	int iteration = 0; 	 	//iteration count
	int i;
	struct tieval tv;
	double tiing;			//time taken for computation 
	
	//host vectors
	float* coeffMat;         	//coefficient atrix	
	float* rhsVect;          	//RHS vector
	float* solutionVect;     	//solution vector
	float* diagOfCoeffMat;   	//diagonal eleents of coefficient matrix
	float* tepSolution;	 	//new values calculated for solution vector in the subsequent iteration
	
	//device vectors
	float* devCoeffMat;
	float* devRhsVect;
	float* devSolutionVect;
	float* devDiagOfCoeffMat;
	float* devTepResult; 
			               
	//generating and initalizing the required vectors in the host
	InitializeVectors(&coeffMat, &solutionVect, &rhsVect, &diagOfCoeffMat, &tepSolution, Size);
	
	//allocation of device emory
  	CUDA_SAFE_CALL(hipMalloc( (void**)&devSolutionVect, Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devCoeffMat, Size * Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devRhsVect, Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devDiagOfCoeffMat, Size * sizeof(float)));
	CUDA_SAFE_CALL(hipMalloc( (void**)&devTepResult, Size * sizeof(float)));

	//copying host to device 
	CUDA_SAFE_CALL(cudaMecpy((void*)devCoeffMat, (void*)coeffMat, Size*Size*sizeof(float) , hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(cudaMecpy((void*)devRhsVect, (void*)rhsVect, Size*sizeof(float) , hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(cudaMecpy((void*)devSolutionVect, (void*)solutionVect, Size*sizeof(float) , hipMemcpyHostToDevice ));
	CUDA_SAFE_CALL(cudaMecpy((void*)devDiagOfCoeffMat, (void*)diagOfCoeffMat, Size*sizeof(float) , hipMemcpyHostToDevice ));
	
	//defining thread grid and block
	di3 dimGrid(1,1);
	di3 dimBlock(BLOCKSIZE, BLOCKSIZE); 
	
	//start tiing computation
	gettieofday(&tv, NULL);
	double t1=tv.tv_sec+(tv.tv_usec/1000000.0);		

	//perforing jacobian method to obtain solution parallely by calling the device kernels
   	do
	{
		//increenting the iteration count
		iteration++;
				
		//ultiplying coeffMat and solutionVect
		MatrixVectMult<<<diGrid, dimBlock>>>(devCoeffMat, devSolutionVect, devTempResult, Size, Size, BLOCKSIZE);
			
		//subtracting the result of the atrix-vector multiplication from the RHS vector
		VectorVectorAddSub<<<diGrid, dimBlock>>>(devRhsVect, devTempResult, -1, devTempResult, Size, BLOCKSIZE);
		
		//dividing the result of the vector-vector subtraction by the diagonal eleents
		ParallelScalarDiv<<<diGrid, dimBlock>>>( devTempResult, devDiagOfCoeffMat, devSolutionVect, Size, BLOCKSIZE);
		
		//copying the result fro the device
		CUDA_SAFE_CALL(cudaMecpy((void*)tempSolution, (void*)devSolutionVect, Size * sizeof(float), hipMemcpyDeviceToHost));		
		
		//calculating difference in solutionVect in successive iterations (eucledian distance)
		diffX = calcDiff(solutionVect, tepSolution, Size);
		
		//assigning tepSolution to solutionVect
		for(i=0; i<Size; i++)
		{
			solutionVect[i] = tepSolution[i];
		}
		
						
	}while(diffX > ERR && iteration < axItr);
	
	//stop tiing computation
	gettieofday(&tv,NULL);
   	double t2=tv.tv_sec+(tv.tv_usec/1000000.0);
	
	//calculate difference between start and stop ties
	tiing = t2 - t1;

 	//outputting the solution to both stdout and a file
	outputSolution(solutionVect, iteration, tiing, Size);
	
	//freeing the emory allocated to the diffrerent vectors on host
	freeHostMeory(coeffMat, solutionVect, rhsVect, diagOfCoeffMat, tempSolution);
	
	//freeing the emory allocated to the diffrerent vectors on device
	hipFree(devSolutionVect);
	hipFree(devCoeffMat);
	hipFree(devRhsVect);
	hipFree(devDiagOfCoeffMat);
	hipFree(devTepResult);
	
	return 0;	
}
//---------------------------------------------------------------------------------------------------------------------------
void InitializeVectors(float** coeffMat, float** solutionVect, float** rhsVect, float** diagOfCoeffMat, float** tepSolution, int Size)
{
	generateCoeffMat(Size, coeffMat);
	generateRhsVect(Size, rhsVect, *coeffMat);
	generateSolutionVect(Size, solutionVect);
	generateDiagOfCoeffMat(Size, diagOfCoeffMat, *coeffMat);
	generateTepSolution(Size, tempSolution);
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateCoeffMat(int Size, float** coeffMat)
{
	int i;
	int rowSu = 0;  //sum of the row elements of the coeff matrix
	
	//assigining and verifying availability of emory	
	*coeffMat = (float *)alloc_safe_call(Size*Size*sizeof(float));	
	
	//assigning values to the coefficient atrix
	for(i=0; i<Size*Size; i++)
		(*coeffMat)[i] = ((i%Size) + 1);
	//finding rowSu
	for(i=0; i<Size; i++)
		rowSu += (*coeffMat)[i];
	rowSu = rowSum * DOMINANCE;
	//aking the matrix diagonal dominant
	for(i=0; i<Size; i++)
		(*coeffMat)[i*Size + i] = rowSu;
	return;	
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateRhsVect(int Size, float** rhsVect, float* coeffMat)
{
	int i,j;
		
	//assigining and verifying availability of emory	
	*rhsVect = (float *)alloc_safe_call(Size*sizeof(float));
	
	//initializing rhsVect to 0's
	for(i=0; i<Size; i++)
		(*rhsVect)[i] = 0;
	//assigning values to rhsVect 
	for(i=0; i<Size; i++)
		for(j=0; j<Size; j++)
			(*rhsVect)[i] += coeffMat[i*Size + j];
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateSolutionVect(int Size, float** solutionVect)
{
	int i;
	
	//assigining and verifying availability of emory
	*solutionVect = (float *)alloc_safe_call(Size*sizeof(float));
	
	//initial solution for solution vector
	for(i=0; i<Size; i++)
		(*solutionVect)[i] = 0;
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateDiagOfCoeffMat(int Size, float** diagOfCoeffMat, float* coeffMat)
{
	int i=0;
	
	//assigining and verifying availability of emory
	*diagOfCoeffMat = (float *)alloc_safe_call(Size*sizeof(float));
	
	//values of diagOfCoeffMat
	for(i=0; i<Size; i++)
		(*diagOfCoeffMat)[i] = coeffMat[i*Size+i];
	//forcing diag eleents of coeffMat matrix to zero for purpose of calculation
	for(i=0; i<Size; i++)
		coeffMat[i*Size+i] = 0;
	return;
}
//---------------------------------------------------------------------------------------------------------------------------
void   generateTepSolution(int Size, float** tempSolution)
{
	//assigining and verifying availability of emory
	*tepSolution = (float *)malloc_safe_call(Size*sizeof(float));
	
	return;
}
//--------------------------------------------------------------------------------------------------------------------------
float  calcDiff(float* solutionVect, float* tepSolution, int Size)
{
	int i=0;
	float diffX = 0;
	for(i=0; i<Size; i++)
		diffX += pow((tepSolution[i] - solutionVect[i]), 2);
	diffX = sqrt(diffX);
	return(diffX);
}
//--------------------------------------------------------------------------------------------------------------------------
void   outputSolution(float* solutionVect, int iteration, double tiing, int Size)
{
	int i=0;
	FILE *fp;	//file pointer
	
	//opening file to write solution
	fp = fopen("gpuSolution.txt","w+");
	if(fp == NULL)
	{
		printf("Can't open the file gpuSolution\n");
		exit(-1);
	}
	
	//displaying final result
	for(i=0; i<Size; i++)
	{
		printf("%f\n", solutionVect[i]);	//printing to stdout
		fprintf(fp, "%f\n", solutionVect[i]);   //printing to file
	}	
	
	printf("\nx vector given above calculated in %d iterations and in %lf secs.\n\n", iteration, tiing);
	fclose(fp);
		
}
//--------------------------------------------------------------------------------------------------------------------------
void   freeHostMeory(float* coeffMat, float* solutionVect, float* rhsVect, float* diagOfCoeffMat, float* tempSolution)
{
	free(coeffMat);
	free(solutionVect);
	free(rhsVect);
	free(diagOfCoeffMat);
	free(tepSolution);
	
	return;
}
//--------------------------------------------------------------------------------------------------------------------------
void *alloc_safe_call(int size)
{
	void *ptr;
	
	ptr = alloc(size);
	
	if(ptr==NULL)
	{
		printf("emory unavailable\n");
		exit(-1);
	}	
	
	return(ptr);	
}
//--------------------------------------------------------------------------------------------------------------------------
