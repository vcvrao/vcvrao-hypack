

/**************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

Example     :  multipleKernels-multiGPU-streams-matrix-matrix-comp.cu

Objective  : The objective is to demonstrate use of CUDA Synchronous
            and CUDA  Asynchronous APIs with CUDA streams for simple
            addition of two nonsquare matrices & compare the execution time
            on multiGPU system.

            Matrix-Matrix Addition kernel is domonstrated

Input       : Number of kernels(optional, default is set to 16)

Output      : Execution-Type(Syn,Asyn),Execution Time in sec Relative-Error

Created     : August-2013

 E-mail     : hpcfte@cdac.in     

********************************************************************************/

/*  inclusion of header file that contains necessary declarions */
#include<pthread.h>
#include <stdio.h>
#include<hip/hip_runtime.h>

#include<time.h>
#include<math.h>
#include<assert.h>

#define EPS 1.0e-14 /* threshhold aprrox epsilion value */
#define BLOCK_SIZE 8
#define NUMROWS 128
#define NUMCOLS 64

int numOfDevicesAvailable;
long int hA, wA, hB, wB ,size;             //holds height and width for MatrixA and MatrixB
double *hAddMatMatA , *hAddMatMatB, *hAddMatMatC;  // holds host matrices
int nkernels;                   // holds total number of concurrent kernels


/* function prototypes */
double matMatAddCheckResult (double *hAddMatMatA,double *hAddMatMatB,double *output,long int numRows,long int numCols);
void memoryAlloc(long int hA, long int wA,long int hB, long int wB);


/* Macro to check for correctness of CUDA API */
#define CUDA_SAFE_CALL(call){\
	hipError_t err = call;                                                    \
    if( hipSuccess != err) {                                                \
        fprintf(stderr, "Cuda error in file '%s' in line %i : %s.\n",        \
                __FILE__, __LINE__, hipGetErrorString( err) );              \
        exit(-1);                                                  \
    }}\


/*
 * Fill in the matrix/vector with double precision values
 */
void fillInData(double* vec,int size)
{
        int ind;
        for(ind=0;ind<size;ind++)
                vec[ind]=drand48() ;
}

/*
*check mem error
*/
void memError(char *arrayname, char *benchmark, int len, char *type)
{

        printf("\nMemory not sufficient to allocate for array %s\n\tBenchmark : %s  \n\tMemory requested = %d number of %s elements\n",arrayname, benchmark, len, type);
        printf("\n\tAborting\n\n");
        exit(-1);
}

/*
*checl grid and block dimensions
*/
void checkBlockGridDim(hipDeviceProp_t devProp,dim3 blockDim,dim3 gridDim)
{

        if( blockDim.x >= devProp.maxThreadsDim[0] || blockDim.y >= devProp.maxThreadsDim[1] || blockDim.z >= devProp.maxThreadsDim[2] )
        {
                printf("\nBlock Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxThreadsDim[0],devProp.maxThreadsDim[1],devProp.maxThreadsDim[2]);
               exit(-1);
        }

        if( gridDim.x >= devProp.maxGridSize[0] || gridDim.y >= devProp.maxGridSize[1] || gridDim.z >= devProp.maxGridSize[2] )
        {
                printf("\nGrid Dimensions exceed the maximum limits:%d * %d * %d \n",devProp.maxGridSize[0],devProp.maxGridSize[1],devProp.maxGridSize[2]);
               exit(-1);
        }
}


/*****************************************
* Matrix Matrix Addition
******************************************/
/* __global__ void kernelMatMatAdd(double *dInMatA, double *dInMatB,double *dInMatC,  int matRowColSize, int threadDim)
  {
	int tidx = threadIdx.x;
    	int tidy = threadIdx.y;
   	int tindex = (threadDim * tidx) + tidy;      // get thread index
    	int maxNumThread = threadDim * threadDim;
    	int pass = 0;
    	int rowCount ;
    	int curColInd ;

    	while( (curColInd = (tindex + maxNumThread * pass))  < matRowColSize )
     	{
        	for( rowCount = 0; rowCount < matRowColSize; rowCount++)
		{
          		 dInMatC[curColInd * matRowColSize + rowCount] = dInMatA[curColInd * matRowColSize + rowCount] + dInMatB[curColInd * matRowColSize + rowCount];
		}
        	pass++;       // move to next column
      	} 

     	__syncthreads();

  } end of Mat Mat Add device code */


__global__ void kernelMatMatAdd(double *dInMatA, double *dInMatB,double *dInMatC,  long int matRowSize, long int matColSize ,int threadDim)
  {
        int tidx = threadIdx.x;
        int tidy = threadIdx.y;
        int tindex = (threadDim * tidx) + tidy;      // get thread index
        int maxNumThread = threadDim * threadDim;
        int pass = 0;
        int rowCount ;
        int curColInd ;

        while( (curColInd = (tindex + maxNumThread * pass))  < matColSize )
        {
                for( rowCount = 0; rowCount < matRowSize; rowCount++)
                {
                         dInMatC[curColInd * matRowSize + rowCount] = dInMatA[curColInd * matRowSize + rowCount] + dInMatB[curColInd * matRowSize + rowCount];
                }
                pass++;       // move to next column
        }

        __syncthreads();

  }/* end of Mat Mat Add device code */



/***************************************************************
function to implement concurrent kernel execution 
***************************************************************/
void funcAsynchConcurrentExec(double *dAddMatMatA, double *dAddMatMatB, double *dAddMatMatC,double *hAddMatMatA, double *hAddMatMatB, double *hAddMatMatC, int nkernels, int NSTREAM, hipStream_t *stream , long int hA, long int wA, long int hB, long int wB,hipDeviceProp_t deviceProp)
{
    	float elapsedTime;           // holds timing variables
    	hipError_t err;               // holds error value

    	/* create CUDA event handles */

	hipEvent_t startEvent, stopEvent;
	CUDA_SAFE_CALL( hipEventCreate(&startEvent));
        CUDA_SAFE_CALL( hipEventCreate(&stopEvent));

	
	/* get all errors before kernel launch */
        if ( err=hipGetLastError())
        {
                printf(" File : %s , Line : %d , Error : %s \n",__FILE__, __LINE__, hipGetErrorString(err));
        }

        /* define blocks and grids check grid and block dimension*/
        dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE); //threads per block
        dim3 dimGrid(1,1); //blocks per grid
        checkBlockGridDim(deviceProp,dimGrid, dimBlock);

        /* Asynchronous kernel execution */
	hipEventRecord(startEvent);
	for( int i=0; i<nkernels; ++i)
	{
        	/* mem copy from host to device asynchcronously */
        	CUDA_SAFE_CALL( hipMemcpyAsync(dAddMatMatA, hAddMatMatA, hA*wA*sizeof(double), hipMemcpyHostToDevice,stream[i]));
        	CUDA_SAFE_CALL( hipMemcpyAsync(dAddMatMatB, hAddMatMatB, hB*wB*sizeof(double), hipMemcpyHostToDevice, stream[i]));
        	CUDA_SAFE_CALL( hipMemcpyAsync(dAddMatMatC, hAddMatMatC, hA*wB*sizeof(double), hipMemcpyHostToDevice, stream[i]));
	}
	for( int i=0; i<nkernels; ++i)
	{
                // queue nkernels  and record when they are done
                //kernelMatMatAdd<<<dimGrid, dimBlock, 0, stream[i]>>>(dAddMatMatA,dAddMatMatB, dAddMatMatC, SIZE,BLOCK_SIZE);
                kernelMatMatAdd<<<dimGrid, dimBlock, 0, stream[i]>>>(dAddMatMatA,dAddMatMatB, dAddMatMatC, NUMROWS,NUMCOLS,BLOCK_SIZE);
        
	}

	for( int i=0; i<nkernels; ++i)
	{
        	/* copy output from device to host */
        	CUDA_SAFE_CALL( hipMemcpyAsync(hAddMatMatC, dAddMatMatC, hA*wB*sizeof(double), hipMemcpyDeviceToHost, stream[i]));
	}
       	CUDA_SAFE_CALL( hipEventRecord(stopEvent));
        CUDA_SAFE_CALL( hipEventSynchronize(stopEvent));
       	CUDA_SAFE_CALL( hipEventElapsedTime(&elapsedTime, startEvent, stopEvent));
        
	/* get all errors from kernel launch */
        if ( err=hipGetLastError())
        {
                printf(" File : %s , Line : %d , Error : %s \n",__FILE__, __LINE__, hipGetErrorString(err));
        }


        /* calculate measured time and gflops */
	double tsecGpu;
        tsecGpu = (double) (elapsedTime  * 1.0e-3);          // converting to seconds from milliseconds

        /* check GPU results against CPU results */
        double errorNorm =  matMatAddCheckResult (hAddMatMatA,hAddMatMatB,hAddMatMatC,hA,wB);
        
	/* print output on screen */
        printf("%s\t%f\t %e\t\n","Asynchronous Concurrent Execution",tsecGpu,errorNorm);
	/* relese GPU events */

    	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);	

	
}


/************************************************************************
functions to execute multiple kernels without stream 
************************************************************************/
void funcSynchExec(double *dAddMatMatA, double *dAddMatMatB, double *dAddMatMatC,double *hAddMatMatA, double *hAddMatMatB, double *hAddMatMatC, int nkernels,long int hA, long int wA, long int hB, long int wB, hipDeviceProp_t deviceProp)
{
    	float elapsedTime;             // holds timing variables
    	hipError_t     err;             // holds error value

    	/* create CUDA event handles */
	
	hipEvent_t startEvent, stopEvent;
	CUDA_SAFE_CALL( hipEventCreate(&startEvent));
        CUDA_SAFE_CALL( hipEventCreate(&stopEvent));

        /* get all errors before  kernel launch */
        if ( err=hipGetLastError())
        {
                printf(" File : %s , Line : %d , Error : %s \n",__FILE__, __LINE__, hipGetErrorString(err));
        }

        /* define blocks and grids check grid and block dimension*/
        dim3 dimBlock(BLOCK_SIZE,BLOCK_SIZE); //threads per block
        dim3 dimGrid(1,1); //blocks per grid
         
        checkBlockGridDim(deviceProp,dimGrid, dimBlock);

        /*Synchronous kernel execution */
        hipEventRecord(startEvent, 0);
	for(int i=0;i<nkernels;i++)
	{
        	/* mem copy from host to device asynchcronously */
        	CUDA_SAFE_CALL( hipMemcpy(dAddMatMatA, hAddMatMatA, hA*wA*sizeof(double), hipMemcpyHostToDevice));
        	CUDA_SAFE_CALL( hipMemcpy(dAddMatMatB, hAddMatMatB, hB*wB*sizeof(double), hipMemcpyHostToDevice));
        	CUDA_SAFE_CALL( hipMemcpy(dAddMatMatC, hAddMatMatC, hA*wB*sizeof(double), hipMemcpyHostToDevice));
	}
        for( int i=0; i<nkernels; ++i)
        { 
                // queue nkernels  and record when they are done
                //kernelMatMatAdd<<<dimGrid, dimBlock>>>(dAddMatMatA,dAddMatMatB, dAddMatMatC, SIZE,BLOCK_SIZE);
                kernelMatMatAdd<<<dimGrid, dimBlock>>>(dAddMatMatA,dAddMatMatB, dAddMatMatC, hA,wB,BLOCK_SIZE);
        }
        for( int i=0; i<nkernels; ++i)
	{
        	/* copy output from device to host */
	        CUDA_SAFE_CALL( hipMemcpy(hAddMatMatC, dAddMatMatC, hA*wB*sizeof(double), hipMemcpyDeviceToHost));
	}	
        /* in this sample we just wait until the GPU is done */
        CUDA_SAFE_CALL( hipEventRecord(stopEvent, 0) );
        CUDA_SAFE_CALL( hipEventSynchronize(stopEvent) );
        CUDA_SAFE_CALL( hipEventElapsedTime(&elapsedTime, startEvent, stopEvent) );

        /* get all errors from kernel launch */
        if ( err=hipGetLastError())
        {
                printf(" File : %s , Line : %d , Error : %s \n",__FILE__, __LINE__, hipGetErrorString(err));
        }


        /* calculate measured time and gflops */
        double tsecGpu = (double) (elapsedTime  * 1.0e-3);

        /* check CPU+GPU results against CPU results */
        double errorNorm =  matMatAddCheckResult (hAddMatMatA,hAddMatMatB,hAddMatMatC,hA,wB);
        
	/* print output on the screen */
	printf("%s\t\t\t%f\t %e\t\n","Synchronous Execution",tsecGpu,errorNorm);

	/* release GPU event */
    	hipEventDestroy(startEvent);
	hipEventDestroy(stopEvent);	

}


/************************************************************
function to check the result with sequential result
***************************************************************/
double matMatAddCheckResult (double *hAddMatMatA,double *hAddMatMatB,double *outputGPU,long int numRows,long int numCols)
{

        int j, flag=0;               //Holds flag value
        double *outputCPU;            //Holds sequential resultant output
        double  errorNorm = 0.0;     // HOlds Error norm value
        double  eps=EPS;              
        double  relativeError=0.0;   // Holds relative error

        assert((outputCPU = (double *)malloc( sizeof(double) * numRows*numCols))!=NULL);

        /*sequential Matrix Matrix Addition result*/
        for( j=0 ; j<numRows*numCols  ; j++)
        {
                outputCPU[j]= hAddMatMatA[j] + hAddMatMatB[j];
        }
        /* check opencl result with sequential result*/
        for( j=0 ; j < numRows*numCols  ; j++)
        {
                if (fabs(outputCPU[j]) > fabs(outputGPU[j]))
                        relativeError = fabs((outputCPU[j] - outputGPU[j]) / outputCPU[j]);
                else
                        relativeError = fabs((outputGPU[j] - outputCPU[j]) / outputGPU[j]);

                if (relativeError > eps)
                {
                        if(errorNorm < relativeError)
                        {
                                errorNorm = relativeError;
                                flag=1;
                        }
                }
        }
        if( flag == 1) {

                printf(" \n\t Results verfication : Failed");
                printf(" \n\t Considered machine precision : %e", eps);
                printf(" \n\t Relative Error                  : %e", errorNorm);

        }

        if(flag==0)
        {
        }
        free(outputCPU);
        return errorNorm;
}
/* function to check device properties related to aynchronous execution */
void checkDeviceProperty(hipDeviceProp_t deviceProp)
{	
	//printf("\nDevice Used :\t %s",deviceProp.name);

        if( (deviceProp.concurrentKernels == 0 )) //check concurrent kernel support
        {
                printf("> GPU does not support concurrent kernel execution\n");
                printf("  CUDA kernel runs will be serialized\n");
        }
        if(deviceProp.asyncEngineCount == 0) //check concurrent data transfer support
        {
                printf("GPU does not support concurrent Data transer and overlaping of kernel execution & data transfer\n");
                printf("Mem copy call will be blocking calls\n");
        }
}

/* function to check for device availability */
void checkDeviceAvailability(int id)
{
	hipError_t     err;             // holds error value
	err=hipSetDevice(id);   //change this to set the code to another GPU
        if (err == cudaErrorDevicesUnavailable)
        {
               printf("\ndevice %d Not available\n",id);
               exit(0);
        }
}
/* Function for memory allocation */
void memoryAlloc(long int hA, long int wA, long int hB,long int wB)
{
	int size;
        /* memory allocate to matrices*/
        CUDA_SAFE_CALL( hipHostMalloc((void**)&hAddMatMatA , hA * wA * sizeof(double), hipHostMallocDefault));
        CUDA_SAFE_CALL( hipHostMalloc((void**)&hAddMatMatB , hB * wB * sizeof(double), hipHostMallocDefault));
        CUDA_SAFE_CALL( hipHostMalloc((void**)&hAddMatMatC , hA * wB * sizeof(double), hipHostMallocDefault));

        /* initialize Matrices*/
        fillInData(hAddMatMatA,hA*wA);
        fillInData(hAddMatMatB,hB*wB);
        for(int index = 0; index < hA*wB ; index++)
                hAddMatMatC[index] = 0;

}

/* Function to check command line arguments */
void check_cmdline_arg(int argc,char* argv[])			
{
	switch(argc)
	{
		case 1:
			printf("\n Number of kernels not specified....default value will be taken\n");
                        nkernels = 16;
			break;

		case 2 :
			nkernels = atoi(argv[1]);              // holds total number of concurrent kernels
			if(nkernels==0)
			{	
				printf("\nWrong input....\n");
				printf("\nUsage : <executable> [nkernels].........aborting \n");
				exit(-1);
			}

			if(nkernels > 16)
        		{
                		printf("\n The maximum number of kernel launches that a device can execute concurrently is 16 \n");
                		printf("\n Kernels will may not be executed concurrently...... \n");
        		} 

			break;
		default :
			 printf("\n Invalid options...\n");
			 printf("\n Usage : <./exe> [nKernels] \n");
			 exit(-1);
	}
}

/* Thread function definition */
void* threadWork(int threadId)
{
	double   *dAddMatMatA, *dAddMatMatB, *dAddMatMatC; // holds device matrices
	hipDeviceProp_t deviceProp;
	hipStream_t *stream;           // holds stream array
        int  NSTREAM ,count,size ;                  // holds total number of streams

	NSTREAM = nkernels;

	checkDeviceAvailability(threadId);

	hipSetDevice(threadId);   
        int device;
        hipGetDevice(&device);
        hipGetDeviceProperties(&deviceProp,device);

	 /* call function to check device properties */
        checkDeviceProperty(deviceProp); // function to check device properties
	
	 size = hA * wA * sizeof(double);
        CUDA_SAFE_CALL( hipMalloc((void**) &dAddMatMatA, size));

        /* allocate device memory*/
        size = hB * wB * sizeof(double);
        CUDA_SAFE_CALL( hipMalloc((void**) &dAddMatMatB,size));

        /* allocate device memory*/
        size = hA * wB * sizeof(double);
        CUDA_SAFE_CALL( hipMalloc((void**) &dAddMatMatC,size));
	
	for(count = 0 ; count < NSTREAM; count++)
                stream = (hipStream_t*) malloc(NSTREAM * sizeof(hipStream_t));
        for(count = 0; count< NSTREAM; count++)
                CUDA_SAFE_CALL( hipStreamCreate(&(stream[count])));

	/* print information on the screen */
	printf("\n\tFor device %d : %s\n ",threadId,deviceProp.name);
        printf("\nNumber of kernels :\t %d", nkernels);
        printf("\nNOTE : TIME_SEC includes data transfer time from host to device, device to host and kernel time");
        printf("\n\nExecution-Type\t\t\t\t Time_sec\t Relative-Error\n");
        printf("======================================================================\n");

        /* call function to execute Asynchronous kernels execution */
        funcAsynchConcurrentExec(dAddMatMatA, dAddMatMatB, dAddMatMatC,hAddMatMatA, hAddMatMatB, hAddMatMatC, nkernels, NSTREAM, stream ,hA,wA,hB,wB,deviceProp);

        /* call function to execute  synchronous kernels execution */
        funcSynchExec(dAddMatMatA, dAddMatMatB, dAddMatMatC,hAddMatMatA, hAddMatMatB, hAddMatMatC, nkernels, hA,wA,hB,wB,deviceProp);

        printf("======================================================================\n");
        /*********** Release all resources***************************/

        /* destroy an array of stream handles */
        for(count = 0; count< NSTREAM; count++)
                CUDA_SAFE_CALL( hipStreamDestroy((stream[count])));

        hipFree(dAddMatMatA);
        hipFree(dAddMatMatB);
        hipFree(dAddMatMatC);

	return 0;
}

/*****************************************************************************
                       main function
******************************************************************************/
int main(int argc, char *argv[])
{
	pthread_t *threads;
	int threadCount , threadStatus,numThreads;

	// get number of available devices
	CUDA_SAFE_CALL(hipGetDeviceCount(&numOfDevicesAvailable));

	numThreads=numOfDevicesAvailable;

	int count;


	hA=hB=NUMROWS;
	wA=wB=NUMCOLS;

        count =0;                       // holds counter variables

	check_cmdline_arg(argc,argv);				// function to check command line arguments

      	/* function to allocate Host and Device matrices*/
	memoryAlloc(hA,wA, hB,wB);

	assert(threads = (pthread_t *)malloc(numThreads * sizeof(pthread_t)));   // allocate memory for number of threads

	// call thread function
	for(threadCount = 0 ; threadCount < numThreads ; threadCount++)
	{
	      threadStatus = pthread_create(&threads[threadCount], NULL,  (void *(*) (void *))threadWork, (void *)(threadCount));
	      if(threadStatus)
		{
			printf("Error in creating the thread and the return status is %d \n",threadStatus);
			exit(-1);
		}
	}

	// join threads with main thread
	for(threadCount = 0 ; threadCount < numThreads ; threadCount++)
        {
              threadStatus = pthread_join(threads[threadCount], NULL);
              if(threadStatus)
                {
                        printf("Error in joining the threads and the return status is %d \n",threadStatus);
                        exit(-1);
                }
        }

	hipHostFree(hAddMatMatA);
	hipHostFree(hAddMatMatB);
	hipHostFree(hAddMatMatC);


	return 0;
}
