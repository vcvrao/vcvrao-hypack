
/**************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                         October 15-18, 2013

 Example             : cudaHostAllocTest.cu

 Objective           : Objective is to demonstrate the time taken by hipMalloc 
                         using the pinned host-memory  
 
 Input               : None

 Output              : Time using hipHostAlloc(up,down),                                              
                         copy speed in MB/s(up,down)                                              

 Created             : August-2013

 E-mail              : hpcfte@cdac.in     

*************************************************************************/

#include "cudaSafeCall.h"

#define SIZE    (64*1024*1024)
float cuda_host_alloc_test( int size, bool up ) {
    hipEvent_t     start, stop;
    int             *a, *dev_a;
    float           elapsedTime;

    CUDA_SAFE_CALL( hipEventCreate( &start ) );
    CUDA_SAFE_CALL( hipEventCreate( &stop ) );

    CUDA_SAFE_CALL( hipHostAlloc( (void**)&a,
                                 size * sizeof( *a ),
                                 hipHostMallocDefault ) );
    CUDA_SAFE_CALL( hipMalloc( (void**)&dev_a,
                              size * sizeof( *dev_a ) ) );

     CUDA_SAFE_CALL( hipEventRecord( start, 0 ) );
    for (int i=0; i<100; i++) {
        if (up)
            CUDA_SAFE_CALL( hipMemcpy( dev_a, a,
                                  size * sizeof( *a ),
                                  hipMemcpyHostToDevice ) );
        else
            CUDA_SAFE_CALL( hipMemcpy( a, dev_a,
                                  size * sizeof( *a ),
                                  hipMemcpyDeviceToHost ) );
    }
    CUDA_SAFE_CALL( hipEventRecord( stop, 0 ) );
    CUDA_SAFE_CALL( hipEventSynchronize( stop ) );
    CUDA_SAFE_CALL( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    CUDA_SAFE_CALL( hipHostFree( a ) );
    CUDA_SAFE_CALL( hipFree( dev_a ) );
    CUDA_SAFE_CALL( hipEventDestroy( start ) );
    CUDA_SAFE_CALL( hipEventDestroy( stop ) );

    return elapsedTime;
}

int main( void ) {
    float           elapsedTime;
    float           MB = (float)100*SIZE*sizeof(int)/1024/1024;
    // now try it with hipHostAlloc
    elapsedTime = cuda_host_alloc_test( SIZE, true );
    printf( "Time using hipHostAlloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy up:  %3.1f\n",
            MB/(elapsedTime/1000) );

    elapsedTime = cuda_host_alloc_test( SIZE, false );
    printf( "Time using hipHostAlloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy down:  %3.1f\n",
            MB/(elapsedTime/1000) );
}

