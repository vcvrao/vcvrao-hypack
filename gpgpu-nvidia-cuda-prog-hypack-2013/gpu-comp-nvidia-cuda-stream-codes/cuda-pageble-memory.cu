/*****************************************************************************

		C-DAC Tech Workshop : hyPACK-2013
                           October 15-18, 2013

 Example               : cuda_malloc_test.cu

 Objective             : Objective is to demonstrate the time taken by hipMalloc 
                         using the pageable host-memory  
 
 Input                 : None

 Output                : Time using hipMalloc(up,down),                                              
                          copy speed in MB/s(up,down)                                              

 Created               : August-2013

 E-mail                : hpcfte@cdac.in     


*********************************************************************************/

#include "cudaSafeCall.h"

#define SIZE    (64*1024*1024)


float cuda_malloc_test( int size, bool up ) {
    hipEvent_t     start, stop;
    int             *a, *dev_a;
    float           elapsedTime;

    CUDA_SAFE_CALL( hipEventCreate( &start ) );
    CUDA_SAFE_CALL( hipEventCreate( &stop ) );

    a = (int*)malloc( size * sizeof( *a ) );
    CUDA_HANDLE_NULL( a );
    CUDA_SAFE_CALL( hipMalloc( (void**)&dev_a,
                              size * sizeof( *dev_a ) ) );

    CUDA_SAFE_CALL( hipEventRecord( start, 0 ) );
    for (int i=0; i<100; i++) {
        if (up)
            CUDA_SAFE_CALL( hipMemcpy( dev_a, a,
                                  size * sizeof( *dev_a ),
                                  hipMemcpyHostToDevice ) );
        else
            CUDA_SAFE_CALL( hipMemcpy( a, dev_a,
                                  size * sizeof( *dev_a ),
                                  hipMemcpyDeviceToHost ) );
    }
    CUDA_SAFE_CALL( hipEventRecord( stop, 0 ) );
    CUDA_SAFE_CALL( hipEventSynchronize( stop ) );
    CUDA_SAFE_CALL( hipEventElapsedTime( &elapsedTime,
                                        start, stop ) );

    free( a );
    CUDA_SAFE_CALL( hipFree( dev_a ) );
    CUDA_SAFE_CALL( hipEventDestroy( start ) );
    CUDA_SAFE_CALL( hipEventDestroy( stop ) );

    return elapsedTime;
}



int main( void ) {
    float           elapsedTime;
    float           MB = (float)100*SIZE*sizeof(int)/1024/1024;


    // try it with hipMalloc
    elapsedTime = cuda_malloc_test( SIZE, true );
    printf( "Time using hipMalloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy up:  %3.1f\n",
            MB/(elapsedTime/1000) );

    elapsedTime = cuda_malloc_test( SIZE, false );
    printf( "Time using hipMalloc:  %3.1f ms\n",
            elapsedTime );
    printf( "\tMB/s during copy down:  %3.1f\n",
            MB/(elapsedTime/1000) );
}

